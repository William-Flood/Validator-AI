#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <cstring>

#define THRESHHOLD 20;
#define BLOCKSIZE 512;


extern "C" {

	__declspec(dllexport) int CalculateBlockCount(int firingNeuronCount) {
		unsigned int blockSize = BLOCKSIZE;
		unsigned int firingBlocks = firingNeuronCount / blockSize;
		if (0 != firingNeuronCount % blockSize) {
			firingBlocks++;
		}
		return firingBlocks;
	}
}


__global__ void activateSensory(const int *sensoryGridBlock, int *activeBlock, const int sensoryIndex, const int neuronCount) {
	activeBlock[threadIdx.x + blockIdx.x*neuronCount*blockDim.x] =
		sensoryGridBlock[threadIdx.x + sensoryIndex*neuronCount + blockIdx.x*neuronCount*blockDim.x];
}

//use dim3 to set multidimensional thread count
//dim3 blockDim(xNeuronCount, yNeuronCount, netCount);
//collapse<<<1,blockDim>>>(activationGrid, span);

__global__ void setValues(int *activationGrid, const int width, const int depth) {
	int neuronIndex = threadIdx.x + blockDim.x * blockIdx.x;
	int collapseOn = threadIdx.y;
	int netIndex = threadIdx.z;
	activationGrid[neuronIndex + collapseOn*width + netIndex*width*depth] = 1;
}

__global__ void increment(int *activationGrid, const int width, const int depth, const int size, const int pad_to) {
	int neuronIndex = threadIdx.x + blockDim.x * blockIdx.x;
	int collapseOn = threadIdx.y;
	int netIndex = threadIdx.z;
	int currentIndex = neuronIndex + collapseOn*width + netIndex*width*depth;
	if (size > currentIndex) {
		for (int i = 0; i < pad_to; i++) {
			activationGrid[currentIndex] += 1;
		}
	}
}

//Successively split the shared memory array in half, adding the values from the second half to the first half 
//(eg adding the value at index 15 to index 7, then the value at index 7 to index 3, then the value at index 3 to index 1, 
//and then adding the value at index 1 to index 0 - at the end, the value at index 0 will contain the sum of all the values in the original list)
__device__ void
reduceBlock(volatile int *sdata, int mySum, const unsigned int tid, unsigned int firingNeuronCount, int* intermediates, unsigned int intermediateIndex)
{
	sdata[tid] = mySum;
	__syncthreads();
	int blockSize = blockDim.x;
	if (firingNeuronCount < threadIdx.x + blockSize*blockIdx.x) {
		return;
	}
	// do reduction in shared mem
	int nextSummedIndex = 256;
	int indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 512)
	//{
	//	if (tid < 256)
	//	{
	//		sdata[tid] = mySum = mySum + sdata[tid + 256];
	//	}
	//
	//	__syncthreads();
	//}

	__syncthreads();
	nextSummedIndex = 128;
	indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 256)
	//{
	//	if (tid < 128)
	//	{
	//		sdata[tid] = mySum = mySum + sdata[tid + 128];
	//	}
	//
	//	__syncthreads();
	//}

	__syncthreads();
	nextSummedIndex = 64;
	indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 128)
	//{
	//	if (tid <  64)
	//	{
	//		sdata[tid] = mySum = mySum + sdata[tid + 64];
	//	}
	//
	//	__syncthreads();
	//}

	//if (tid < 32)
	//{
	__syncthreads();
	nextSummedIndex = 32;
	indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 64)
	//{
	//	sdata[tid] = mySum = mySum + sdata[tid + 32];
	//}

	__syncthreads();
	nextSummedIndex = 16;
	indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 32)
	//{
	//	sdata[tid] = mySum = mySum + sdata[tid + 16];
	//}

	nextSummedIndex = 8;
	indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 16)
	//{
	//	sdata[tid] = mySum = mySum + sdata[tid + 8];
	//}

	nextSummedIndex = 4;
	indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 8)
	//{
	//	sdata[tid] = mySum = mySum + sdata[tid + 4];
	//}

	nextSummedIndex = 2;
	indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 4)
	//{
	//	sdata[tid] = mySum = mySum + sdata[tid + 2];
	//}

	nextSummedIndex = 1;
	indexEscape = (tid + nextSummedIndex) < blockSize;
	mySum += indexEscape * sdata[(tid + nextSummedIndex) * indexEscape];
	sdata[tid] = mySum;
	//if (blockSize >= 2)
	//{
	//	sdata[tid] = mySum = mySum + sdata[tid + 1];
	//}
	//}
	//intermediates[intermediateIndex] = sdata[tid];
}

/*template <unsigned int blockSize, bool nIsPow2>
__device__ void
reduceBlocks(const int *g_idata, int *g_odata, unsigned int n)
{
	extern __shared__ int sdata[];

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + threadIdx.x;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	int mySum = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{
		mySum += g_idata[i];

		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n)
			mySum += g_idata[i + blockSize];

		i += gridSize;
	}

	// do reduction in shared mem
	reduceBlock<blockSize>(sdata, mySum, tid);

	// write result for this block to global mem
	if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; }
}*/



__device__ void
tallyNeurons(int* intermediateBlock, int* tallyBlock, int* tempTally, unsigned int firingNeuronCount)
{
	extern __shared__ int sdata[];
	int threshhold = THRESHHOLD;
	// perform first level of reduction,
	// reading from global memory, writing to shared memory

	unsigned int firingNeuron = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int countEscape = firingNeuronCount > firingNeuron;
	
	unsigned int tallyingNeuron = blockIdx.y;
	unsigned int tallyingNeuronCount = gridDim.y;
	unsigned int network = blockIdx.z;
	unsigned int networkIndex = network*tallyingNeuronCount*firingNeuronCount;
	unsigned int firingNeuronIndex = networkIndex + (firingNeuron)*tallyingNeuronCount;
	unsigned int synapseIndex = tallyingNeuron + firingNeuronIndex;
	int mySum = intermediateBlock[synapseIndex] * countEscape * (threshhold <= tallyBlock[(firingNeuron)+network*tallyingNeuronCount]);
	//intermediateBlock[synapseIndex] += 1000* intermediateBlock[synapseIndex] * countEscape * (threshhold <= tallyBlock[(firingNeuron)+network*tallyingNeuronCount]);;
	// do reduction in shared mem
	reduceBlock(sdata, mySum, threadIdx.x, firingNeuronCount, intermediateBlock, synapseIndex);

	// write result for this block to global mem
	unsigned int tempTallyIndex = network*gridDim.x*tallyingNeuronCount + tallyingNeuron*gridDim.x + blockIdx.x;
	if (threadIdx.x == 0) {
		tempTally[tempTallyIndex] = sdata[0];
		//Debug
		//intermediateBlock[synapseIndex] += (sdata[0] + 1) * 1000;
	}

}


__global__ void
AddSensoryTally(int* sensoryBlock, int sensoryCount, int* tallyBlock, int* tempTally, unsigned int sensoryIndex, unsigned int tallyingNeuronCount)
{
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tallyingNeuronID = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int inRangeFlag = tallyingNeuronID < tallyingNeuronCount;
	unsigned int sensoryNeuronIndex = blockIdx.y * tallyingNeuronCount * sensoryCount + sensoryIndex * tallyingNeuronCount;
	unsigned int sensoryValue = sensoryBlock[sensoryNeuronIndex+ tallyingNeuronID] * inRangeFlag;
	//sensoryBlock[sensoryNeuronIndex + tallyingNeuronID] = 1000 * inRangeFlag - 500;
	int* storageRoot = tallyBlock + (tempTally - tallyBlock) * (1 - inRangeFlag);
	unsigned int tallyingNeuronIndex = tallyingNeuronID + blockIdx.y * tallyingNeuronCount;
	storageRoot[tallyingNeuronIndex] += sensoryValue;
}


/*template <unsigned int blockSize, bool nIsPow2>
__global__ void
reduceMultiPass(const int *g_idata, int *g_odata, unsigned int n)
{
	reduceBlocks<blockSize, nIsPow2>(g_idata, g_odata, n);
}*/


/*template <unsigned int blockSize, bool nIsPow2>
__global__ void
reduceMultiPass(const int *g_idata, int *g_odata, unsigned int n, unsigned int sensoryIndex)
{
	reduceBlocks<blockSize, nIsPow2>(g_idata, g_odata, n, sensoryIndex);
}*/




__global__ void
tallyMultiPass(int* intermediateBlock, int* tallyBlock, int* tempTally, unsigned int blockSize)
{
	tallyNeurons(intermediateBlock, tallyBlock, tempTally, blockSize);
}


__global__ void finalizeTally(int* tallyBlock, int* tempTally, int intermediateCount) {
	unsigned int blockSize = BLOCKSIZE;
	int threshhold = THRESHHOLD;
	unsigned int blockCount = intermediateCount / blockSize + (0 != intermediateCount % blockSize);

	//Sets tally at the tallied neuron index at the net index to zero, should it be an intermediate neurorn with a tally greater than the threshhold
	tallyBlock[threadIdx.x + blockDim.x*blockIdx.x] = tallyBlock[threadIdx.x + blockDim.x*blockIdx.x] * 
		(!(threadIdx.x <intermediateCount && threshhold <= tallyBlock[threadIdx.x + blockDim.x*blockIdx.x]));

	int totalDelta = 0;
	int* tempNeuronTally = tempTally + blockDim.x*blockIdx.x * blockCount + threadIdx.x*blockCount;
	for (int i = 0; i < blockCount; i++) {
		totalDelta += tempNeuronTally[i];
	}

	int rawSet = tallyBlock[threadIdx.x + blockDim.x*blockIdx.x] + totalDelta;

	tallyBlock[threadIdx.x + blockDim.x*blockIdx.x] = rawSet * (rawSet > 0);


}

bool isPow2(unsigned int x)
{
	return ((x&(x - 1)) == 0);
}

/*void reduce(int size, int threads, int blocks, int *d_idata, int *d_odata)
{
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

	// choose which of the optimized versions of reduction to launch
	if (isPow2(size))
	{
		switch (threads)
		{
		case 512:
			reduceMultiPass<512, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 256:
			reduceMultiPass<256, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 128:
			reduceMultiPass<128, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 64:
			reduceMultiPass< 64, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 32:
			reduceMultiPass< 32, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 16:
			reduceMultiPass< 16, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  8:
			reduceMultiPass<  8, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  4:
			reduceMultiPass<  4, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  2:
			reduceMultiPass<  2, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  1:
			reduceMultiPass<  1, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;
		}
	}
	else
	{
		switch (threads)
		{
		case 512:
			reduceMultiPass<512, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 256:
			reduceMultiPass<256, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 128:
			reduceMultiPass<128, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 64:
			reduceMultiPass< 64, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 32:
			reduceMultiPass< 32, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 16:
			reduceMultiPass< 16, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  8:
			reduceMultiPass<  8, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  4:
			reduceMultiPass<  4, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  2:
			reduceMultiPass<  2, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  1:
			reduceMultiPass<  1, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;
		}
	}
}
*/


__device__ void tally(int firingNeuronCount, int tallyingNeuronCount, int netCount, int* intermediateBlock, int* tallyBlock, int* tempTally)
{
	unsigned int blockSize = BLOCKSIZE;
	blockSize = firingNeuronCount * (firingNeuronCount <= blockSize) + blockSize * (firingNeuronCount > blockSize);
	unsigned int firingBlocks = firingNeuronCount / blockSize + (0 != firingNeuronCount % blockSize);
	dim3 dimBlock(blockSize, 1, 1);
	dim3 dimGrid(firingBlocks, tallyingNeuronCount, netCount);
	int sharedMemSize = blockSize * sizeof(int);

	tallyMultiPass<< < dimGrid, dimBlock, sharedMemSize>> >(intermediateBlock, tallyBlock, tempTally, firingNeuronCount);

}


__device__ void sensoryTally(int netCount, int* sensoryBlock, unsigned int sensoryCount, int* tallyBlock, int* tempTally, unsigned int sensoryIndex, int tallyingNeuronCount )
{
	unsigned int blockSize = BLOCKSIZE;
	blockSize = tallyingNeuronCount * (tallyingNeuronCount <= blockSize) + blockSize * (blockSize < tallyingNeuronCount);
	unsigned int tallyingBlockCount = tallyingNeuronCount / blockSize + (0 != tallyingNeuronCount % blockSize);
	dim3 dimGrid(tallyingBlockCount, netCount);

	AddSensoryTally << < dimGrid, blockSize>> >(sensoryBlock, sensoryCount, tallyBlock, tempTally, sensoryIndex, tallyingNeuronCount);

}

__global__ void cudaRunCycle(int sensoryCount, int intermediateCount, int motorCount, int netCount, int* sensoryBlock, int* intermediateBlock, int* tallyBlock, int* tempTally, int* sensoryInput, int sensoryInputLength) {
	for (int i = 0; i < sensoryInputLength; i++) {
		int sensoryIndex = *(sensoryInput + i);
		int tallyingNeuronCount = intermediateCount + motorCount;
		sensoryTally(netCount, sensoryBlock, sensoryCount, tallyBlock, tempTally, sensoryIndex, tallyingNeuronCount);
		for (int insideLoop = 0; insideLoop < 10; insideLoop++) {
			tally(intermediateCount, tallyingNeuronCount, netCount, intermediateBlock, tallyBlock, tempTally);
			finalizeTally << <netCount, tallyingNeuronCount >> >(tallyBlock, tempTally, intermediateCount);
		}
	}
}


extern "C"
{

	/*__declspec(dllexport) int* testCollapse(int size, int pad_to) {
		int *toCollapse = (int*)malloc(size *sizeof(int));
		int *g_toCollapse;
		hipMalloc(&g_toCollapse, size * sizeof(int));
		int *g_collapseResult;
		hipMalloc(&g_collapseResult, size * sizeof(int));

		int threads, blocks;
		if (size < 512) {
			threads = size / 4;
			if ((size & 3) == 0) {
				blocks = 4;
			}
			else {
				blocks = 5;
			}
		}
		else {
			threads = 512;
			if (size % 512 == 0) {
				blocks = size / 512;
			}
			else {
				blocks = size / 512 + 1;
			}
		}
		dim3 blockDim(threads, 1, 1);
		increment <<<blocks, blockDim >>>(g_toCollapse, 1, 1,size,pad_to);
		
		
		reduce(size,threads,blocks,g_toCollapse,g_collapseResult);
		

		hipMemcpy(toCollapse, g_collapseResult, size * sizeof(int), hipMemcpyDeviceToHost);
		hipFree(g_toCollapse);
		hipFree(g_collapseResult);
		return toCollapse;
	}*/




	__declspec(dllexport) int blockSize() {
		return BLOCKSIZE;
	}

	__declspec(dllexport) void release(int* toRelease) {
		free(toRelease);
	}

	__declspec(dllexport) void cuda_release(int* toRelease) {
		hipFree(toRelease);
	}

	//int synapseListLength = (intermediateCount + motorCount)*netCount;
	//int sensoryBlockSize = sensoryCount*synapseListLength;
	//int intermediateBlockSize = intermediateCount;
	//int tallyGridSize = (intermediateCount + motorCount)*netCount;
	//int activeBlockSize = synapseListLength*(intermediateCount + 1)*netCount;
	//int totalSize = sensoryBlockSize + intermediateBlockSize + tallyGridSize + activeBlockSize;
	__declspec(dllexport) int* declare_transfer_block(int totalSize) {
		int* transfer_pointer = (int*)malloc(totalSize*sizeof(int));
		hipError_t err = hipGetLastError();

		if (hipSuccess != err)
		{
			const char * errorString = hipGetErrorString(err);
		}
		return transfer_pointer;


	}

	__declspec(dllexport) int* establish_net_block(int* toEstablish, int totalSize) {
		int* net_block;
		int memSize = totalSize * sizeof(int);
		hipMalloc(&net_block, memSize);

		hipMemcpy(net_block, toEstablish, memSize,hipMemcpyHostToDevice);
		return net_block;
	}

	__declspec(dllexport) int* findIntermediateBlock(int* sensoryBlock, int sensoryCount, int intermediateCount, int motorCount, int netCount) {
		int sensoryBlockSize = sensoryCount*(intermediateCount + motorCount) * netCount;
		return sensoryBlock + sensoryBlockSize;
	}

	__declspec(dllexport) int* findTallyBlock(int* intermediateBlock, int intermediateCount, int motorCount, int netCount) {
		int intermediateSize = intermediateCount*(intermediateCount + motorCount)*netCount;
		return intermediateBlock + intermediateSize;
	}

	__declspec(dllexport) int* findTempTallyBlock(int* tallyBlock, int intermediateCount, int motorCount, int netCount) {
		int tallySize = (intermediateCount + motorCount)*netCount;
		return tallyBlock + tallySize;
	}

	__declspec(dllexport) void runCycle(int sensoryCount, int intermediateCount, int motorCount, int netCount, int* sensoryBlock, int* intermediateBlock, int* tallyBlock, int* tempTally, int* sensoryInput, int sensoryInputLength) {
		cudaRunCycle << <1, 1 >> > (sensoryCount, intermediateCount, motorCount, netCount, sensoryBlock, intermediateBlock, tallyBlock, tempTally, sensoryInput, sensoryInputLength);
		
		const char * errorString;
		hipError_t err = hipGetLastError();

		if (hipSuccess != err)
		{
			errorString = hipGetErrorString(err);
		}
		/*int firingNeuronCount = intermediateCount+1;
		int tallyingNeuronCount = intermediateCount + motorCount;
		tally(firingNeuronCount, tallyingNeuronCount, netCount, sensoryBlock, intermediateBlock, tallyBlock, tempTally, sensoryIndex, sensoryCount);
		firingNeuronCount = intermediateCount;
		for (int insideLoop = 0; insideLoop < 10; insideLoop++) {
			tally(firingNeuronCount, tallyingNeuronCount, netCount, intermediateBlock, tallyBlock, tempTally);
			finalizeTally << <netCount, tallyingNeuronCount >> >(tallyBlock, tempTally, intermediateCount, firingNeuronCount);
		}*/

	}

	__declspec(dllexport) int* getTally(int* tallyBlock, int tallyingNeuronCount) {
		int* tallyReport = (int*)malloc(tallyingNeuronCount*sizeof(int));
		hipMemcpy(tallyReport, tallyBlock, tallyingNeuronCount * sizeof(int), hipMemcpyDeviceToHost);
		return tallyReport;
	}

	__declspec(dllexport) int* getNet(int* encodedNet, int netSize) {
		int* netReport = (int*)malloc(netSize * sizeof(int));
		hipMemcpy(netReport, encodedNet, netSize * sizeof(int), hipMemcpyDeviceToHost);
		return netReport;
	}


	__declspec(dllexport) int* establish_sensory_input(int* toEstablish, int totalSize) {
		int* net_block;
		int memSize = totalSize * sizeof(int);
		hipMalloc(&net_block, memSize);

		hipMemcpy(net_block, toEstablish, memSize, hipMemcpyHostToDevice);
		return net_block;
	}
}
